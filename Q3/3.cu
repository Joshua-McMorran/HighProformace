
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__device__ char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}


//comapres 2 strings and returns 0 if both strings are the same
__device__ int stringCompare(char * stringOne, char * stringTwo, int stringLength){
	
	int stringCount = 0;	
	for(int i = 0; i <stringLength; i++){
		if(stringOne[i] != stringTwo[i]){
				stringCount++;
		}
	}
	return stringCount;
}

__global__ void crack(char * alphabet, char * numbers, char * originalEncrypted, char * foundPassword){

	char genRawPass[4];
	char * encrypted; 
	
	genRawPass[0] = alphabet[blockIdx.x];
	genRawPass[1] = alphabet[blockIdx.y];

	genRawPass[2] = numbers[threadIdx.x];
	genRawPass[3] = numbers[threadIdx.y];
	
	encrypted = CudaCrypt(genRawPass); 
	
	//printf("genpass = %s encrypted = %s\n", genRawPass, encrypted);
	
	if(stringCompare(originalEncrypted, encrypted, 11) == 0){
		//printf("Password found encrypted = %s genpass = %s\n", encrypted, genRawPass);
		
		//sets the found password to the created empty char array on the GPU
		//*foundPassword = *genRawPass;  
		for(int i=0; i <4; i++){
			foundPassword[i] = genRawPass[i];
			//printf("The found 1 password is -> %s\n", foundPassword);
			printf("char 1 = %i char 2 = %i char 3 = %i char 4 = %i char 5 = %i\n",foundPassword[0], foundPassword[1], foundPassword[2],foundPassword[3], foundPassword[4]);
		}
		printf("found password 2 = %s\n", foundPassword);
	}
}

int main (int argc, char* argv[]){
	/*  encrypted examples:
		rnqdwy5134 pz38
		lhkuoq8453 jr61
		iehdwy3191 gz15
	*/

	char alphabet[26] = { 'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z' };
	char numbers[10] = { '0', '1', '2', '3', '4', '5', '6' ,'7', '8', '9' };
	
	char * originalEncrypted = "rnqdwy5134";
	 
	
	//creating GPU variable and copying it
	char * gpuAlphabet;
	hipMalloc((void**) &gpuAlphabet, sizeof(char) * 26); 
	hipMemcpy(gpuAlphabet, alphabet, sizeof(char) * 26, hipMemcpyHostToDevice);
	
	char * gpuNumbers;
	hipMalloc((void**) &gpuNumbers, sizeof(char) * 10);
	hipMemcpy(gpuNumbers, numbers, sizeof(char) * 10, hipMemcpyHostToDevice);
	
	char *gpuOriginalEncrypted;
	hipMalloc((void**) &gpuOriginalEncrypted, sizeof(char) * 11);
	hipMemcpy(gpuOriginalEncrypted, originalEncrypted, sizeof(char) * 11, hipMemcpyHostToDevice);
	
	//creates an empty char array allowing for the pass to be set on GPU
	char *gpuFoundPassword;
	hipMalloc((void**) &gpuFoundPassword, sizeof(char) * 4);

	crack<<< dim3(26,26,1), dim3(10,10,1) >>>(gpuAlphabet, gpuNumbers, gpuOriginalEncrypted, gpuFoundPassword);
	hipDeviceSynchronize();
	
	//copys the now set array from the GPU  back to the CPU
	char * foundPassword = (char*)malloc(sizeof(char)*4);
	hipMemcpy(foundPassword, gpuFoundPassword, sizeof(char) * 4, hipMemcpyDeviceToHost);
	
	
	printf("The found 3 password is -> '%c%c%c%c'\n", foundPassword[0],foundPassword[1],foundPassword[2],foundPassword[3]);
	
	//print char 1 at a time as 4 times for the 4 chars 
	
	//free all memory
	hipFree(gpuAlphabet);
	hipFree(gpuNumbers);
	hipFree(gpuOriginalEncrypted);
	hipFree(gpuFoundPassword);
	free(foundPassword);

} 
