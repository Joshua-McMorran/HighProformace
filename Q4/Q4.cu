#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

__global__ void blurImage(unsigned char * inputImage, unsigned char * outputImage, int width, int height){


	int uniqueID = blockDim.x * blockIdx.x + threadIdx.x; //to create unique ID for each pixel
	
	//printf("%d \n", uniqueID);

	int row = uniqueID / width;
	int col = uniqueID % width;
	
	//printf("width2 = %d height2 = %d \n", width, height);

	//printf("UID = %d col = %d row = %d \n",uniqueID ,col, row);	
	
	//targetPixel = the pixel that the current thread is bluring
	int targetPixel = uniqueID * 4;
	
	
	//check if row is equal to 0, if equal to 0 then you are on the top of the image	
	int topBorder = 0;
	if(row == 0){
		topBorder = 1;
	}
	
	//check if row is equal to height - 1, if row is equal then you are on the bottom of the image
	int bottomBorder = 0;
	if(row == height - 1){
		bottomBorder = 1;
	}
	
	//check if col is equal to 0, if equal to 0 then you are at the left border of the image
	int westBorder = 0;
	if(col == 0){
		westBorder = 1;
	}
	
	//check if col is equal to 0, if equal to 0 then you are at the right border of the image
	int eastBorder = 0;
	if( col == width - 1){
		eastBorder = 1;
	}
	
	//this sets the RGBT values of the target pixel access the array and assign the colour to the appropriate value
	int r = inputImage[targetPixel+0];
	int g = inputImage[targetPixel+1];
	int b = inputImage[targetPixel+2];
	int t = inputImage[targetPixel+3];

	//printf("UID = %d t = %d r = %d \n", uniqueID, t ,r);
	
	//used to declare North red, South red, West red, East red etc
	int Nr = 0, Ng = 0, Nb = 0, Nt = 0;
	int Sr = 0, Sg = 0, Sb = 0, St = 0;
	int Wr = 0, Wg = 0, Wb = 0, Wt = 0;
	int Er = 0, Eg = 0, Eb = 0, Et = 0;
	
	//used to declare diaginal NE, NW etc
	int NEr = 0, NEg = 0, NEb = 0, NEt = 0;
	int NWr = 0, NWg = 0, NWb = 0, NWt = 0;
	int SWr = 0, SWg = 0, SWb = 0, SWt = 0;
	int SEr = 0, SEg = 0, SEb = 0, SEt = 0;
	
	
	//used for the math to average the pixel (blur)
	int pixelCount = 1;
	
	//check used to see if the primary pixel is on the top of the image
	if(topBorder != 1){
		//not on the top border therefore pixels above current row
		
		int northPixelIndex = ((row - 1)* width * 4) + (col * 4);
		//printf("northPixelIndex = %d\n", northPixelIndex);
		
		/* 	(row - 1) = 0 * 3 = 0 * 4 = 0 (col = 1 * 4) = 4 
			total is 4th index in array which is pixel[1] value R
			
			this is the algroithm to find the north pixel of the target pixel
					(x-1)*y*4+(z*4);
		*/
				
		Nr = inputImage[northPixelIndex];
		Ng = inputImage[northPixelIndex + 1];
		Nb = inputImage[northPixelIndex + 2];
		Nt = inputImage[northPixelIndex + 3];
		
		pixelCount++;
		
		//printf("UID = %d Nr = %d Ng = %d Nb = %d Nt = %d\n",uniqueID ,Nr,Ng,Nb,Nt);
		
			//northEast - NE
			if(eastBorder != 1){
				
				int NEpixelIndex = ((row - 1)* width * 4) + ((col + 1) * 4);
				
				NEr = inputImage[NEpixelIndex];
				NEg = inputImage[NEpixelIndex + 1];
				NEb = inputImage[NEpixelIndex + 2];
				NEt = inputImage[NEpixelIndex + 3];
				
				pixelCount++;
			}
			
			if(westBorder !=1 ){
				
				int NWpixelIndex = ((row - 1)* width * 4) + ((col - 1) * 4);
				
				NWr = inputImage[NWpixelIndex];
				NWg = inputImage[NWpixelIndex + 1];
				NWb = inputImage[NWpixelIndex + 2];
				NWt = inputImage[NWpixelIndex + 3];
				
				pixelCount++;
			
			}
		
	}
	
	if(bottomBorder != 1){
		
		int southPixelIndex = ((row + 1)* width * 4) + (col * 4);
		
		Sr = inputImage[southPixelIndex];
		Sg = inputImage[southPixelIndex + 1];
		Sb = inputImage[southPixelIndex + 2];
		St = inputImage[southPixelIndex + 3];
		
		pixelCount++;
		
			if(westBorder != 1){
				
				int SWPixelIndex = ((row + 1)* width * 4) + ((col - 1) * 4);
		
				SWr = inputImage[SWPixelIndex];
				SWg = inputImage[SWPixelIndex + 1];
				SWb = inputImage[SWPixelIndex + 2];
				SWt = inputImage[SWPixelIndex + 3];
				
				pixelCount++;
			}
			
			if(eastBorder != 1){
				
				int SEPixelIndex = ((row + 1)* width * 4) + ((col + 1)* 4);
				
				SEr = inputImage[SEPixelIndex];
				SEg = inputImage[SEPixelIndex + 1];
				SEb = inputImage[SEPixelIndex + 2];
				SEt = inputImage[SEPixelIndex + 3];
				
				pixelCount++;
			}
	}
	
	if(westBorder != 1){
			
		int westPixelIndex = (row * width * 4) + ((col - 1) * 4);
		
		Wr = inputImage[westPixelIndex];
		Wg = inputImage[westPixelIndex + 1];
		Wb = inputImage[westPixelIndex + 2];
		Wt = inputImage[westPixelIndex + 3];
		
		pixelCount++;
	}
	
	if(eastBorder != 1){
		
		int eastPixelIndex = (row * width * 4) + ((col + 1) *4);
		
		Er = inputImage[eastPixelIndex];
		Eg = inputImage[eastPixelIndex + 1];
		Eb = inputImage[eastPixelIndex + 2];
		Et = inputImage[eastPixelIndex + 3];
		
		pixelCount++;
	}
	
	
	int sumR = r + Nr + Sr + Er + Wr + NWr + NEr + SEr + SWr;
	int sumG = g + Ng + Sg + Eg + Wg + NWg + NEg + SEg + SWg;
	int sumB = b + Nb + Sb + Eb+ Wb + NWb + NEb + SEb + SWb;
	
	//printf("UID = %d sumR = %d sumG = %d sumB = %d \n", uniqueID, sumR, sumG, sumB);
	//printf("Uid = %d pixelCount = %d \n", uniqueID, pixelCount);
	
	int averageR = sumR / pixelCount;
	int averageG = sumG / pixelCount;
	int averageB = sumB / pixelCount;
	int averageT = t;
	
	//printf("UID = %d averageR = %d averageG = %d averageB = %d pixelCount = %d \n",uniqueID, averageR, averageG, averageB, pixelCount);
	//this assignes the newImage with the average pixel values creating the original image blurred
	outputImage[targetPixel] = averageR;
	outputImage[targetPixel + 1] = averageG;
	outputImage[targetPixel + 2] = averageB;
	outputImage[targetPixel + 3] = averageT;
	
	
}


int main(int argc, char ** argv){
	
	unsigned char* cpuImage; //stores the image data on the CPU	
	unsigned int errorDecode; //varible will hold whether there was issues loading the PNG image
	unsigned  int width, height; //stores the width and heught of the image	
	
	char * filename = argv[1];
	char * newFileName = argv[2];
	
	errorDecode = lodepng_decode32_file(&cpuImage, &width, &height, filename);
	if(errorDecode){
	printf("error %u: %s\n", errorDecode, lodepng_error_text(errorDecode));
	}
	
	printf("width = %d height = %d \n", width, height);
	
	int arrayImageSize = width*height*4; //Store number accurate to the size of the array needed
	int memorySize = arrayImageSize * sizeof(unsigned char); //Store memory size needed in variable+
	
	unsigned char cpuOutImage[arrayImageSize]; //used to store the array size needed to create the image
	
	unsigned char* gpuInput;
	unsigned char* gpuOutput;
	
	hipMalloc((void**) &gpuInput, memorySize);
	hipMalloc((void**) &gpuOutput, memorySize);
	
	hipMemcpy(gpuInput, cpuImage, memorySize, hipMemcpyHostToDevice);
	
	//if width = 3 and height = 3, blueImage will run 9 times (3x3)
	blurImage <<< dim3(width, 1, 1),dim3(height, 1, 1) >>> (gpuInput, gpuOutput, width, height);
	hipDeviceSynchronize();
	
	hipMemcpy(cpuOutImage, gpuOutput, memorySize, hipMemcpyDeviceToHost);
	
	unsigned int errorEncode = lodepng_encode32_file(newFileName, cpuOutImage, width, height);
	if(errorEncode) {
	printf("error %u: %s\n", errorEncode, lodepng_error_text(errorEncode));
	}
	hipFree(gpuInput);
	hipFree(gpuOutput);
}
